#include "hip/hip_runtime.h"

#include <external_dependency.h>
#include <stdio.h>

__global__ void squared_kernel(int *in, int *out) {

  for (unsigned int i=0;i<blockDim.x;++i) {
    // /*const*/ unsigned int thread = threadIdx.x;
    out[threadIdx.x] = in[threadIdx.x] * in[threadIdx.x];
  }
};

void squared(int* in, int* out, int dim) {
  // Setup kernel problem size
  dim3 blocksize(dim,1,1);
  dim3 gridsize(1,1,1);

  // Call kernel
  squared_kernel<<<gridsize, blocksize>>>(in, out);
}

int doit()
{
  hipFree(0);
  CHECK_CUDA_ERROR();

  int h_val[DIM];
  int h_result[DIM];

  for(int i = 0; i < DIM; ++i)
    h_val[i] = i;

  // Allocate device memory
  unsigned int size = sizeof(int) * DIM;
  int* d_val;
  hipMalloc((void**)&d_val, size);
  CHECK_CUDA_ERROR();

  int* d_result;
  hipMalloc((void**)&d_result, size);
  CHECK_CUDA_ERROR();

  // Send input to device
  hipMemcpy(d_val, h_val, size, hipMemcpyHostToDevice);
  CHECK_CUDA_ERROR();

  // Call the kernel wrapper
  squared(d_val, d_result, DIM);
  CHECK_CUDA_ERROR();

  // Get back results
  hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost);
  CHECK_CUDA_ERROR();

  for(int i = 0; i < DIM; ++i)
    printf("%d ^ 2 = %d\n", h_val[i], h_result[i]);

  // Free memory
  hipFree((void*)d_val);
  CHECK_CUDA_ERROR();

  hipFree((void*)d_result);
  CHECK_CUDA_ERROR();

  return 0;
}


